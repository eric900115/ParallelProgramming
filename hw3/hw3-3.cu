#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

//======================
#define DEV_NO 0
hipDeviceProp_t prop;

#define BlockingFactor 64
#define MAX_NUM_THREAD 1024 // Maximum nuber of threads per block
#define MEM_y_Offest 16 // 1024 / 64
#define Half_BF 32

// V : # of Vertex
// E : # of Edge
// n : dimension of Dist is n x n (n is used for padding the original graph) 
int n, V, E;
int* Dist;

const int INF = (1 << 30) - 1;

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&V, sizeof(int), 1, file);
    fread(&E, sizeof(int), 1, file);

    if(V % 256 == 0) {
        n = V;
    }
    else {
        n = (V / 256 + 1) * 256;
    }

    Dist = (int*)malloc(n * n * sizeof(int));

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j) {
                Dist[i * n + j] = 0;
            } else {
                Dist[i * n + j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < E; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0] * n + pair[1]] = pair[2];
    }

    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (Dist[i * n + j] >= INF) Dist[i * n + j] = INF;
        }
    }

    if(n == V) {
        fwrite(Dist, sizeof(int), V * V, outfile);
    }
    else {
        for(int i = 0; i < V; i++) {
            fwrite(Dist + i * n, sizeof(int), V, outfile);
        }
    }

    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }

__global__ void blockedFW_Phase1(int *D, unsigned int round, unsigned int V) {

    int x = threadIdx.x;
    int y = threadIdx.y;

    // the address of required data in global memory
    int global_x = x + round * BlockingFactor;
    int global_y = y + round * BlockingFactor;

    // Cache the current data block being calculated
    __shared__ int shared_D[BlockingFactor][BlockingFactor];

    // load data from golbal memory to shared memory
    shared_D[y][x] = D[global_y * V + global_x];
    shared_D[y + Half_BF][x] = D[(global_y + Half_BF) * V + global_x];
    shared_D[y][x + Half_BF] = D[global_y * V + global_x + Half_BF];
    shared_D[y + Half_BF][x + Half_BF] = D[(global_y + Half_BF) * V + global_x + Half_BF];

    __syncthreads();

    // execute phase 1 of Blocked FW
    #pragma unroll 64
    for(int i = 0; i < BlockingFactor; i++) {
        shared_D[y][x] = min(shared_D[y][x],
            shared_D[y][i] + shared_D[i][x]);

        shared_D[y + Half_BF][x] = min(shared_D[y + Half_BF][x], 
            shared_D[y + Half_BF][i] + shared_D[i][x]);

        shared_D[y][x + Half_BF] = min(shared_D[y][x + Half_BF],
            shared_D[y][i] + shared_D[i][x + Half_BF]);

        shared_D[y + Half_BF][x + Half_BF] = min(shared_D[y + Half_BF][x + Half_BF],
            shared_D[y + Half_BF][i] + shared_D[i][x + Half_BF]);

        __syncthreads();
    }

    // store data from shared mem to global mem
    D[global_y * V + global_x] = shared_D[y][x];
    D[(global_y + Half_BF) * V + global_x] = shared_D[y + Half_BF][x];
    D[global_y * V + global_x + Half_BF] = shared_D[y][x + Half_BF];
    D[(global_y + Half_BF) * V + global_x + Half_BF] = shared_D[y + Half_BF][x + Half_BF];
}

__global__ void blockedFW_Phase2(int *D, unsigned int round, unsigned int V) {

    if(blockIdx.y == round)
        return;

    int x = threadIdx.x;
    int y = threadIdx.y;

    int global_x, global_y;
    int col_x, col_y, row_x, row_y;
    int blockID = blockIdx.y;

    __shared__ int shared_Pivot_Block[BlockingFactor][BlockingFactor];
    __shared__ int shared_Pivot_Col[BlockingFactor][BlockingFactor];
    __shared__ int shared_Pivot_Row[BlockingFactor][BlockingFactor];

    // calculate memory access offset
    global_x = x + round * BlockingFactor;
    global_y = y + round * BlockingFactor;
    col_x = x + round * BlockingFactor;
    col_y = y + blockID * BlockingFactor;
    row_x = x + blockID * BlockingFactor;
    row_y = y + round * BlockingFactor;

    // load data to shared memory
    shared_Pivot_Block[y][x] = D[global_y * V + global_x];
    shared_Pivot_Block[y][x + Half_BF] = D[global_y * V + global_x + Half_BF];
    shared_Pivot_Block[y + Half_BF][x] = D[(global_y + Half_BF) * V + global_x];
    shared_Pivot_Block[y + Half_BF][x + Half_BF] = D[(global_y + Half_BF) * V + global_x + Half_BF];

    shared_Pivot_Col[y][x] = D[col_y * V + col_x];
    shared_Pivot_Col[y][x + Half_BF] = D[col_y * V + col_x + Half_BF];
    shared_Pivot_Col[y + Half_BF][x] = D[(col_y + Half_BF) * V + col_x];
    shared_Pivot_Col[y + Half_BF][x + Half_BF] = D[(col_y + Half_BF) * V + col_x + Half_BF];

    shared_Pivot_Row[y][x] = D[row_y * V + row_x];
    shared_Pivot_Row[y][x + Half_BF] = D[row_y * V + row_x + Half_BF];
    shared_Pivot_Row[y + Half_BF][x] = D[(row_y + Half_BF) * V + row_x];
    shared_Pivot_Row[y + Half_BF][x + Half_BF] = D[(row_y + Half_BF) * V + row_x + Half_BF];

    __syncthreads();

    // Calculate Blocked Folyd Warshall
    #pragma unroll 64
    for(int k = 0; k < BlockingFactor; k++) {

        // Calculate Pivot Column
        shared_Pivot_Col[y][x] = min(shared_Pivot_Col[y][x], 
            shared_Pivot_Col[y][k] + shared_Pivot_Block[k][x]);

        shared_Pivot_Col[y][x + Half_BF] = min(shared_Pivot_Col[y][x + Half_BF], 
            shared_Pivot_Col[y][k] + shared_Pivot_Block[k][x + Half_BF]);

        shared_Pivot_Col[y + Half_BF][x] = min(shared_Pivot_Col[y + Half_BF][x], 
            shared_Pivot_Col[y + Half_BF][k] + shared_Pivot_Block[k][x]);

        shared_Pivot_Col[y + Half_BF][x + Half_BF] = min(shared_Pivot_Col[y + Half_BF][x + Half_BF], 
            shared_Pivot_Col[y + Half_BF][k] + shared_Pivot_Block[k][x + Half_BF]);

        // Calculate Pivot Row
        shared_Pivot_Row[y][x] = min(shared_Pivot_Row[y][x], 
            shared_Pivot_Block[y][k] + shared_Pivot_Row[k][x]);

        shared_Pivot_Row[y][x + Half_BF] = min(shared_Pivot_Row[y][x + Half_BF], 
            shared_Pivot_Block[y][k] + shared_Pivot_Row[k][x + Half_BF]);

        shared_Pivot_Row[y + Half_BF][x] = min(shared_Pivot_Row[y + Half_BF][x], 
            shared_Pivot_Block[y + Half_BF][k] + shared_Pivot_Row[k][x]);

        shared_Pivot_Row[y + Half_BF][x + Half_BF] = min(shared_Pivot_Row[y + Half_BF][x + Half_BF], 
            shared_Pivot_Block[y + Half_BF][k] + shared_Pivot_Row[k][x + Half_BF]);

    }

    // store data to global memory
    D[col_y * V + col_x] = shared_Pivot_Col[y][x];
    D[col_y * V + col_x + Half_BF] = shared_Pivot_Col[y][x + Half_BF];
    D[(col_y + Half_BF) * V + col_x] = shared_Pivot_Col[y + Half_BF][x];
    D[(col_y + Half_BF) * V + col_x + Half_BF] = shared_Pivot_Col[y + Half_BF][x + Half_BF];

    D[row_y * V + row_x] = shared_Pivot_Row[y][x];
    D[row_y * V + row_x + Half_BF] = shared_Pivot_Row[y][x + Half_BF];
    D[(row_y + Half_BF) * V + row_x] = shared_Pivot_Row[y + Half_BF][x];
    D[(row_y + Half_BF) * V + row_x + Half_BF] = shared_Pivot_Row[y + Half_BF][x + Half_BF];
}

__global__ void blockedFW_Phase3(int *D, unsigned int round, unsigned int V, unsigned int offset) {

    // process block(y, x)
    // block(y, x) depends on block(y, round) and block(round, x)

    if((blockIdx.x == round) || (blockIdx.y + offset == round))
        return;

    int data_0, data_1, data_2, data_3;
    int x, y;
    int global_x, global_y;
    int col_x, col_y, row_x, row_y;

    __shared__ int shared_Pivot_Col[BlockingFactor][BlockingFactor];
    __shared__ int shared_Pivot_Row[BlockingFactor][BlockingFactor];

    // calculate memory access offset
    x = threadIdx.x;
    y = threadIdx.y;
    global_x = x + blockIdx.x * BlockingFactor;
    global_y = y + (blockIdx.y + offset) * BlockingFactor;
    col_x = x + round * BlockingFactor;
    col_y = y + (blockIdx.y + + offset) * BlockingFactor;
    row_x = x + blockIdx.x * BlockingFactor;
    row_y = y + round * BlockingFactor;

    // load data to non global memory
    shared_Pivot_Col[y][x] = D[col_y * V + col_x];
    shared_Pivot_Col[y][x + Half_BF] = D[col_y * V + col_x + Half_BF];
    shared_Pivot_Col[y + Half_BF][x] = D[(col_y + Half_BF) * V + col_x];
    shared_Pivot_Col[y + Half_BF][x + Half_BF] = D[(col_y + Half_BF) * V + col_x + Half_BF];

    shared_Pivot_Row[y][x] = D[row_y * V + row_x];
    shared_Pivot_Row[y][x + Half_BF] = D[row_y * V + row_x + Half_BF];
    shared_Pivot_Row[y + Half_BF][x] = D[(row_y + Half_BF) * V + row_x];
    shared_Pivot_Row[y + Half_BF][x + Half_BF] = D[(row_y + Half_BF) * V + row_x + Half_BF];

    __syncthreads();

    data_0 = D[global_y * V + global_x];
    data_1 = D[(global_y + Half_BF) * V + global_x];
    data_2 = D[global_y * V + global_x + Half_BF];
    data_3 = D[(global_y + Half_BF) * V + global_x + Half_BF];

    // calculation of Blocked FW
    #pragma unroll 64
    for(int k = 0; k < BlockingFactor; k++) {
        data_0 = min(data_0, 
            shared_Pivot_Col[y][k] + shared_Pivot_Row[k][x]);
        data_1 = min(data_1, 
            shared_Pivot_Col[y + Half_BF][k] + shared_Pivot_Row[k][x]);
        data_2 = min(data_2, 
            shared_Pivot_Col[y][k] + shared_Pivot_Row[k][x + Half_BF]);
        data_3 = min(data_3, 
            shared_Pivot_Col[y + Half_BF][k] + shared_Pivot_Row[k][x + Half_BF]);
    }

    // store data to global memory
    D[global_y * V + global_x] = data_0;
    D[(global_y + Half_BF) * V + global_x] = data_1;
    D[global_y * V + global_x + Half_BF] = data_2;
    D[(global_y + Half_BF) * V + global_x + Half_BF] = data_3;
}

int main(int argc, char* argv[]) {

    input(argv[1]);

    int *device_dist[2];

    #pragma omp parallel num_threads(2)
    {
        unsigned int offset;

        unsigned int cpu_thread_id = omp_get_thread_num();

        hipSetDevice(cpu_thread_id);

        hipMalloc((void **) &device_dist[cpu_thread_id], n * n * sizeof(int));

        hipMemcpy(device_dist[cpu_thread_id], Dist, n * n * sizeof(int), hipMemcpyHostToDevice);

        // decide to use how many blocks and threads
        dim3 blockNum1(1, 1);
        dim3 blockNum2(1, n / BlockingFactor);
        dim3 blockNum3(n / BlockingFactor, n / BlockingFactor / 2);
        dim3 threadNum(32, 32);

        if(cpu_thread_id == 0) {
            offset = 0;
        }
        else {
            int num_blocks = n / BlockingFactor;
            offset = num_blocks / 2;
            if(num_blocks % 2 != 0) {
                blockNum3.y += 1;
            }
        }

        // execute blocked FW on device
        for(int round = 0; round < (n / BlockingFactor); round++) {
            blockedFW_Phase1 <<<blockNum1, threadNum>>> (device_dist[cpu_thread_id], round, n);
            blockedFW_Phase2 <<<blockNum2, threadNum>>> (device_dist[cpu_thread_id], round, n);
            blockedFW_Phase3 <<<blockNum3, threadNum>>> (device_dist[cpu_thread_id], round, n, offset);

            #pragma omp barrier

            if(((round + 1) >= offset) && ((round + 1) < (offset + blockNum3.y))) {
                if(cpu_thread_id == 0) {
                    int addr_offset = (round + 1) * BlockingFactor * n;
                    hipMemcpyPeer(device_dist[1] + addr_offset, 1, device_dist[0] + addr_offset, 0, BlockingFactor * n * sizeof(int));
                }
                else {
                    int addr_offset = (round + 1) * BlockingFactor * n;
                    hipMemcpyPeer(device_dist[0] + addr_offset, 0, device_dist[1] + addr_offset, 1, BlockingFactor * n * sizeof(int));
                }
            }

            #pragma omp barrier
        }

        if(cpu_thread_id == 0) {
            hipMemcpy(Dist, device_dist[0], BlockingFactor * blockNum3.y * n * sizeof(int), hipMemcpyDeviceToHost);
        }
        else {
            int addr_offset = offset * BlockingFactor * n;
            hipMemcpy(Dist + addr_offset, device_dist[1] + addr_offset, BlockingFactor * blockNum3.y * n * sizeof(int), hipMemcpyDeviceToHost);
        }

        #pragma omp barrier

        hipFree(device_dist[cpu_thread_id]);
    }

    // output data
    output(argv[2]);

    /*for(int i = 0; i < V; i++) {
        for(int j = 0; j < V; j++)
            printf("%d ", Dist[i * n + j]);
        printf("\n");
    }*/

    return 0;
}